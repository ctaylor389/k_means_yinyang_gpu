#include "hip/hip_runtime.h"
#include "kmeansGPU.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
 if (code != hipSuccess)
 {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
 }
}

/////////////////////////////////////////////
// Host functions for calling CUDA kernels //
/////////////////////////////////////////////

double startFullOnGPU(PointInfo *pointInfo,
                    CentInfo *centInfo,
                    DTYPE *pointData,
                    DTYPE *centData,
                    const int numPnt,
                    const int numCent,
                    const int numGrp,
                    const int numDim,
                    const int maxIter,
                    unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;
  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, numGrp);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);


  DTYPE *devMaxDriftArr = NULL;
  hipMalloc(&devMaxDriftArr, sizeof(DTYPE) * numGrp);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentData = NULL;
  hipMalloc(&devOldCentData, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                        sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       numGrp,
                                       numDim);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                           sizeof(unsigned int),hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDriftArr,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

    assignPointsFull<<<NBLOCKS, BLOCKSIZE, grpLclSize>>>(devPointInfo,devCentInfo,
                                                         devPointData,devPointLwrs,
                                                         devCentData,devMaxDriftArr,
                                                         numPnt,numCent,numGrp,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,devConFlag,numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr,
        devConFlag, sizeof(unsigned int),
                    hipMemcpyDeviceToHost));
  }
  // calc final centroids (for matching results with lloyds)
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDriftArr,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDriftArr);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);

  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}


double startSimpleOnGPU(PointInfo *pointInfo,
                      CentInfo *centInfo,
                      DTYPE *pointData,
                      DTYPE *centData,
                      const int numPnt,
                      const int numCent,
                      const int numGrp,
                      const int numDim,
                      const int maxIter,
                      unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;
  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);


  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, numGrp);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  DTYPE *devMaxDriftArr = NULL;
  hipMalloc(&devMaxDriftArr, sizeof(DTYPE) * numGrp);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);
  
  DTYPE *devOldCentData = NULL;
  hipMalloc(&devOldCentData, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       numGrp,
                                       numDim);

  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                           sizeof(unsigned int),hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);


    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDriftArr,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

    assignPointsSimple<<<NBLOCKS, BLOCKSIZE, grpLclSize>>>(devPointInfo,devCentInfo,
                                                           devPointData,devPointLwrs,
                                                           devCentData,devMaxDriftArr,
                                                           numPnt,numCent,numGrp,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                         devConFlag,
                                         numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr, devConFlag,
                           sizeof(unsigned int), hipMemcpyDeviceToHost));
  }
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDriftArr,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                       sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,
                       sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDriftArr);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);

  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}


double startSimpleOnGPU(PointInfo *pointInfo,
                      CentInfo *centInfo,
                      DTYPE *pointData,
                      DTYPE *centData,
                      const int numPnt,
                      const int numCent,
                      const int numGrp,
                      const int numDim,
                      const int maxIter,
                      const int numGPU,
                      unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  int numPnts[numGPU];
  for (int i = 0; i < numGPU; i++)
  {
    if (numPnt % numGPU != 0 && i == numGPU-1)
    {
      numPnts[i] = (numPnt / numGPU) + (numPnt % numGPU);
    }

    else
    {
      numPnts[i] = numPnt / numGPU;
    }

  }

  // variable initialization
  unsigned int hostConFlagArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hostConFlagArr[i] = 1;
  }

  unsigned int *hostConFlagPtrArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hostConFlagPtrArr[i] = &hostConFlagArr[i];
  }

  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo[numGPU];
  DTYPE *devPointData[numGPU];
  DTYPE *devPointLwrs[numGPU];

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hipSetDevice(i);

    // alloc dataset to GPU
    gpuErrchk(hipMalloc(&devPointInfo[i], sizeof(PointInfo)*(numPnts[i])));

    // copy input data to GPU
    gpuErrchk(hipMemcpy(devPointInfo[i],
                         pointInfo+(i*numPnt/numGPU),
                         (numPnts[i])*sizeof(PointInfo),
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&devPointData[i], sizeof(DTYPE) * numPnts[i] * numDim));

    gpuErrchk(hipMemcpy(devPointData[i],
                         pointData+((i*numPnt/numGPU) * numDim),
                         sizeof(DTYPE)*numPnts[i]*numDim,
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&devPointLwrs[i], sizeof(DTYPE) * numPnts[i] *
                         numGrp));

    gpuErrchk(hipMemcpy(devPointLwrs[i],
                         pointLwrs+((i*numPnt/numGPU) * numGrp),
                         sizeof(DTYPE)*numPnts[i]*numGrp,
                         hipMemcpyHostToDevice));
  }

  // store centroids on device
  CentInfo *devCentInfo[numGPU];
  DTYPE *devCentData[numGPU];
  DTYPE *devOldCentData[numGPU];

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));

    // alloc dataset and drift array to GPU
    gpuErrchk(hipMalloc(&devCentInfo[i], sizeof(CentInfo)*numCent));
    
    // alloc the old position data structure
    gpuErrchk(hipMalloc(&devOldCentData[i], sizeof(DTYPE) * numDim * numCent));

    // copy input data to GPU
    gpuErrchk(hipMemcpy(devCentInfo[i],
                         centInfo, sizeof(CentInfo)*numCent,
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&devCentData[i], sizeof(DTYPE)*numCent*numDim));
    gpuErrchk(hipMemcpy(devCentData[i],
                        centData, sizeof(DTYPE)*numCent*numDim,
                        hipMemcpyHostToDevice));
  }

  DTYPE *devMaxDriftArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devMaxDriftArr[i], sizeof(DTYPE) * numGrp);
  }

  // centroid calculation data
  DTYPE *devNewCentSum[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devNewCentSum[i], sizeof(DTYPE) * numCent * numDim);
  }

  DTYPE *devOldCentSum[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devOldCentSum[i], sizeof(DTYPE) * numCent * numDim);
  }

  unsigned int *devNewCentCount[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devNewCentCount[i], sizeof(unsigned int) * numCent);
  }

  unsigned int *devOldCentCount[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devOldCentCount[i], sizeof(unsigned int) * numCent);
  }

  unsigned int *devConFlagArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devConFlagArr[i], sizeof(unsigned int));
    gpuErrchk(hipMemcpy(devConFlagArr[i],
              hostConFlagPtrArr[i], sizeof(unsigned int),
              hipMemcpyHostToDevice));
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum[i],
                                              devOldCentSum[i],
                                              devNewCentCount[i],
                                              devOldCentCount[i],
                                              numCent,
                                              numDim);

  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr[i], numGrp);
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    // do single run of naive kmeans for initial centroid assignments
    initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo[i],
                                         devCentInfo[i],
                                         devPointData[i],
                                         devPointLwrs[i],
                                         devCentData[i],
                                         numPnts[i],
                                         numCent,
                                         numGrp,
                                         numDim);
  }

  CentInfo **allCentInfo = (CentInfo **)malloc(sizeof(CentInfo*)*numGPU);
  for (int i = 0; i < numGPU; i++)
  {
    allCentInfo[i] = (CentInfo *)malloc(sizeof(CentInfo)*numCent);
  }

  DTYPE **allCentData = (DTYPE **)malloc(sizeof(DTYPE*)*numGPU);
  for (int i = 0; i < numGPU; i++)
  {
    allCentData[i] = (DTYPE *)malloc(sizeof(DTYPE)*numCent*numDim);
  }

  CentInfo *newCentInfo = (CentInfo *)malloc(sizeof(CentInfo) * numCent);

  DTYPE *newCentData = (DTYPE *)malloc(sizeof(DTYPE) * numCent * numDim);
  for (int i = 0; i < numCent; i++)
  {
    for (int j = 0; j < numDim; j++)
    {
      newCentData[(i * numDim) + j] = 0;
    }
  }

  DTYPE *oldCentData = (DTYPE *)malloc(sizeof(DTYPE) * numCent * numDim);

  DTYPE *newMaxDriftArr;
  newMaxDriftArr=(DTYPE *)malloc(sizeof(DTYPE)*numGrp);
  for (int i = 0; i < numGrp; i++)
  {
    newMaxDriftArr[i] = 0.0;
  }

  unsigned int doesNotConverge = 1;

  // loop until convergence
  while(doesNotConverge && index < maxIter)
  {
    doesNotConverge = 0;

    for (int i = 0; i < numCent; i++)
    {
      newCentInfo[i].count = 0;
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      hostConFlagArr[i] = 0;
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(devConFlagArr[i],
                hostConFlagPtrArr[i], sizeof(unsigned int),
                hipMemcpyHostToDevice));
    }

    // clear maintained data on device
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr[i], numGrp);
    }


    // calculate data necessary to make new centroids
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                         devPointData[i],devOldCentSum[i],
                                         devNewCentSum[i],devOldCentCount[i],
                                         devNewCentCount[i],numPnts[i],numDim);

    }

    // make new centroids
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                             devCentData[i],devOldCentData[i],
                                             devOldCentSum[i],devNewCentSum[i],
                                             devMaxDriftArr[i],devOldCentCount[i],
                                             devNewCentCount[i],numCent,numDim);

    }

    if (numGPU > 1)
    {
      for (int i = 0; i < numGrp; i++)
      {
        newMaxDriftArr[i] = 0.0;
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
        gpuErrchk(hipSetDevice(i));
        gpuErrchk(hipMemcpy(allCentInfo[i],
                            devCentInfo[i], sizeof(CentInfo)*numCent,
                            hipMemcpyDeviceToHost));
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
        gpuErrchk(hipSetDevice(i));
        gpuErrchk(hipMemcpy(allCentData[i],
                            devCentData[i], sizeof(DTYPE)*numCent*numDim,
                            hipMemcpyDeviceToHost));
      }

      calcWeightedMeans(newCentInfo, allCentInfo, newCentData, oldCentData,
        allCentData, newMaxDriftArr, numCent, numGrp, numDim, numGPU);

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
          gpuErrchk(hipSetDevice(i));

          // copy input data to GPU
          gpuErrchk(hipMemcpy(devCentInfo[i],
                      newCentInfo, sizeof(cent)*numCent,
                                  hipMemcpyHostToDevice));
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
          gpuErrchk(hipSetDevice(i));

          // copy input data to GPU
          gpuErrchk(hipMemcpy(devCentData[i],
                      newCentData, sizeof(DTYPE)*numCent*numDim,
                                  hipMemcpyHostToDevice));
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
          gpuErrchk(hipSetDevice(i));
          gpuErrchk(hipMemcpy(devMaxDriftArr[i],
                       newMaxDriftArr, numGrp*sizeof(DTYPE),
                                  hipMemcpyHostToDevice));
      }
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      assignPointsSimple<<<NBLOCKS,BLOCKSIZE,grpLclSize>>>(devPointInfo[i],
                                                           devCentInfo[i],
                                                           devPointData[i],
                                                           devPointLwrs[i],
                                                           devCentData[i],
                                                           devMaxDriftArr[i],
                                                           numPnts[i],numCent,
                                                           numGrp,numDim);
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo[i],
                                           devConFlagArr[i],
                                           numPnts[i]);

    }

    index++;

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(hostConFlagPtrArr[i],
          devConFlagArr[i], sizeof(unsigned int),
                      hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < numGPU; i++)
    {
      if (hostConFlagArr[i])
      {
        doesNotConverge = 1;
      }
    }
  }

  // calculate data necessary to make new centroids
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                        devPointData[i],devOldCentSum[i],
                                        devNewCentSum[i],devOldCentCount[i],
                                        devNewCentCount[i],numPnts[i],numDim);
  }

  // make new centroids
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                             devCentData[i],devOldCentData[i],
                                             devOldCentSum[i],devNewCentSum[i],
                                             devMaxDriftArr[i],devOldCentCount[i],
                                             devNewCentCount[i],numCent,numDim);
  }

  if (numGPU > 1)
  {
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(allCentInfo[i],
                          devCentInfo[i], sizeof(CentInfo)*numCent,
                          hipMemcpyDeviceToHost));
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(allCentData[i],
                          devCentData[i], sizeof(DTYPE)*numCent*numDim,
                          hipMemcpyDeviceToHost));
    }

    calcWeightedMeans(newCentInfo, allCentInfo, newCentData, oldCentData,
      allCentData, newMaxDriftArr, numCent, numGrp, numDim, numGPU);

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
        gpuErrchk(hipSetDevice(i));

        // copy input data to GPU
        gpuErrchk(hipMemcpy(devCentInfo[i],
                    newCentInfo, sizeof(cent)*numCent,
                                hipMemcpyHostToDevice));
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
        gpuErrchk(hipSetDevice(i));

        // copy input data to GPU
        gpuErrchk(hipMemcpy(devCentData[i],
                    newCentData, sizeof(DTYPE)*numCent*numDim,
                                hipMemcpyHostToDevice));
    }
  }

  hipDeviceSynchronize();

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));

    // copy finished clusters and points from device to host
    gpuErrchk(hipMemcpy(pointInfo+((i*numPnt/numGPU)),
                devPointInfo[i], sizeof(PointInfo)*numPnts[i], hipMemcpyDeviceToHost));
  }

  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData, devCentData[0],
                       sizeof(DTYPE)*numCent*numDim,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  for (int i = 0; i < numGPU; i++)
  {
    hipFree(devPointInfo[i]);
    hipFree(devPointData[i]);
    hipFree(devPointLwrs[i]);
    hipFree(devCentInfo[i]);
    hipFree(devCentData[i]);
    hipFree(devMaxDriftArr[i]);
    hipFree(devNewCentSum[i]);
    hipFree(devOldCentSum[i]);
    hipFree(devNewCentCount[i]);
    hipFree(devOldCentCount[i]);
    hipFree(devConFlagArr[i]);
  }

  free(allCentInfo);
  free(allCentData);
  free(newCentInfo);
  free(newCentData);
  free(oldCentData);
  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}

void calcWeightedMeans(CentInfo *newCentInfo,
                       CentInfo **allCentInfo,
                       DTYPE *newCentData,
                       DTYPE *oldCentData,
                       DTYPE **allCentData,
                       DTYPE *newMaxDriftArr,
                       const int numCent,
                       const int numGrp,
                       const int numDim,
                       const int numGPU)
{
  DTYPE numerator = 0;
  DTYPE denominator = 0;
  DTYPE zeroNumerator = 0;
  int zeroCount = 0;

  for (int i = 0; i < numCent; i++)
  {
      for (int j = 0; j < numDim; j++)
      {
          oldCentData[(i * numDim) + j] = newCentData[(i * numDim) + j];
      }
  }

  for (int i = 0; i < numGPU; i++)
  {
      for (int j = 0; j < numCent; j++)
      {
        newCentInfo[j].count += allCentInfo[i][j].count;

        newCentInfo[j].groupNum = allCentInfo[0][j].groupNum;
      }
  }

  for (int j = 0; j < numCent; j++)
  {
      for (int k = 0; k < numDim; k++)
      {
          for (int l = 0; l < numGPU; l++)
          {
              if (allCentInfo[l][j].count == 0)
              {
                  zeroCount++;
                  zeroNumerator += allCentData[l][(j * numDim) + k];
              }

              numerator +=
              allCentData[l][(j * numDim) + k]*allCentInfo[l][j].count;

              denominator += allCentInfo[l][j].count;
          }

          if (denominator != 0)
          {
              newCentData[(j * numDim) + k] = numerator/denominator;
          }

          else
          {
              newCentData[(j * numDim) + k] = zeroNumerator/zeroCount;
          }

          zeroCount = 0;
          zeroNumerator = 0;
          numerator = 0;
          denominator = 0;
      }

      newCentInfo[j].drift = calcDisCPU(&newCentData[j*numDim],
                                           &oldCentData[j*numDim],
                                           numDim);

      if (newCentInfo[j].drift > newMaxDriftArr[newCentInfo[j].groupNum])
        {
          newMaxDriftArr[newCentInfo[j].groupNum] = newCentInfo[j].drift;
        }
  }
}

double startSuperOnGPU(PointInfo *pointInfo,
                     CentInfo *centInfo,
                     DTYPE *pointData,
                     DTYPE *centData,
                     const int numPnt,
                     const int numCent,
                     const int numDim,
                     const int maxIter,
                     unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);


  // group centroids
  for(int j = 0; j < numCent; j++)
  {
    centInfo[j].groupNum = 0;
  }

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt);
  for(int i = 0; i < numPnt; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, 1);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  DTYPE *devMaxDrift = NULL;
  hipMalloc(&devMaxDrift, sizeof(DTYPE));

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);
  
  DTYPE *devOldCentData = NULL;
  hipMalloc(&devOldCentData, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                         sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDrift, 1);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       1,
                                       numDim);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                           sizeof(unsigned int), hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDrift, 1);

    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDrift,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

    assignPointsSuper<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                              devPointData,devPointLwrs,
                                              devCentData,devMaxDrift,
                                              numPnt,numCent,1,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                         devConFlag,
                                         numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr, devConFlag,
                         sizeof(unsigned int), hipMemcpyDeviceToHost));
  }
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDrift,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                         sizeof(PointInfo)*numPnt, hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData, devCentData,
                         sizeof(DTYPE)*numDim*numCent, hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDrift);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);

  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}

double startLloydOnGPU(PointInfo *pointInfo,
                     CentInfo *centInfo,
                     DTYPE *pointData,
                     DTYPE *centData,
                     const int numPnt,
                     const int numCent,
                     const int numDim,
                     const int maxIter,
                     unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;

  int index = 0;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                         sizeof(unsigned int), hipMemcpyHostToDevice));


  clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                                 devNewCentCount,
                                                 numCent,
                                                 numDim);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                         sizeof(unsigned int),hipMemcpyHostToDevice));

    assignPointsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                              devCentInfo,
                                              devPointData,
                                              devCentData,
                                              numPnt,
                                              numCent,
                                              numDim);

    clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                                   devNewCentCount,
                                                   numCent,
                                                   numDim);
    // calculate data necessary to make new centroids
    calcCentDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                              devPointData,
                                              devNewCentSum,
                                              devNewCentCount,
                                              numPnt,
                                              numDim);

    // make new centroids
    calcNewCentroidsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                                  devCentInfo,
                                                  devCentData,
                                                  devNewCentSum,
                                                  devNewCentCount,
                                                  numCent,
                                                  numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,devConFlag,numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr,devConFlag,
                           sizeof(unsigned int),hipMemcpyDeviceToHost));
  }
  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                         sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,
                         sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devNewCentSum);
  hipFree(devNewCentCount);
  hipFree(devConFlag);

  endTime = omp_get_wtime();
  return endTime - startTime;
}

PointInfo *storePointInfoOnGPU(PointInfo *pointInfo,
                               const int numPnt)
{
  PointInfo *devPointInfo = NULL;
  gpuErrchk(hipMalloc(&devPointInfo, sizeof(PointInfo)*numPnt));
  gpuErrchk(hipMemcpy(devPointInfo, pointInfo,
                         sizeof(PointInfo)*numPnt, hipMemcpyHostToDevice));
  return devPointInfo;
}

CentInfo *storeCentInfoOnGPU(CentInfo *centInfo,
                             const int numCent)
{
  CentInfo *devCentInfo = NULL;
  gpuErrchk(hipMalloc(&devCentInfo, sizeof(CentInfo) * numCent));
  gpuErrchk(hipMemcpy(devCentInfo, centInfo,
                         sizeof(CentInfo)*numCent, hipMemcpyHostToDevice));
  return devCentInfo;
}

DTYPE *storeDataOnGPU(DTYPE *data,
                      const int numVec,
                      const int numFeat)
{
  DTYPE *devData = NULL;
  gpuErrchk(hipMalloc(&devData, sizeof(DTYPE) * numVec * numFeat));
  gpuErrchk(hipMemcpy(devData, data,
                         sizeof(DTYPE)*numVec*numFeat, hipMemcpyHostToDevice));
  return devData;
}

void warmupGPU()
{
  unsigned int *dev_tmp;
  unsigned int *tmp;
  tmp = (unsigned int*)malloc(sizeof(unsigned int));
  *tmp = 0;
  hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));

  warmup<<<1,256>>>(dev_tmp);

  hipMemcpy(tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(dev_tmp);
}

void warmupGPU(const int numGPU)
{
  for (int i = 0; i < numGPU; i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
}


// distcalc overloads

double startFullOnGPU(PointInfo *pointInfo,
                    CentInfo *centInfo,
                    DTYPE *pointData,
                    DTYPE *centData,
                    const int numPnt,
                    const int numCent,
                    const int numGrp,
                    const int numDim,
                    const int maxIter,
                    unsigned int *ranIter,
                    unsigned long long int *countPtr)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;
  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, numGrp);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);


  DTYPE *devMaxDriftArr = NULL;
  hipMalloc(&devMaxDriftArr, sizeof(DTYPE) * numGrp);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentData = NULL;
  hipMalloc(&devOldCentData, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                        sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       numGrp,
                                       numDim,
                                       countPtr);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                           sizeof(unsigned int),hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDriftArr,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

    assignPointsFull<<<NBLOCKS, BLOCKSIZE, grpLclSize>>>(devPointInfo,devCentInfo,
                                                         devPointData,devPointLwrs,
                                                         devCentData,devMaxDriftArr,
                                                         numPnt,numCent,numGrp,
                                                         numDim, countPtr);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,devConFlag,numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr,
        devConFlag, sizeof(unsigned int),
                    hipMemcpyDeviceToHost));
  }
  // calc final centroids (for matching results with lloyds)
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDriftArr,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDriftArr);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);

  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}


double startSimpleOnGPU(PointInfo *pointInfo,
                      CentInfo *centInfo,
                      DTYPE *pointData,
                      DTYPE *centData,
                      const int numPnt,
                      const int numCent,
                      const int numGrp,
                      const int numDim,
                      const int maxIter,
                      const int numGPU,
                      unsigned int *ranIter,
                      unsigned long long int *countPtr)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  int numPnts[numGPU];
  for (int i = 0; i < numGPU; i++)
  {
    if (numPnt % numGPU != 0 && i == numGPU-1)
    {
      numPnts[i] = (numPnt / numGPU) + (numPnt % numGPU);
    }
    else
    {
      numPnts[i] = numPnt / numGPU;
    }
  }
  unsigned long long int hostDistCalc = 0;
  unsigned long long int *hostDistCalcCount = &hostDistCalc;

  unsigned long long int *hostDistCalcCountArr;
  hostDistCalcCountArr=(unsigned long long int *)malloc(sizeof(unsigned long long int)*numGPU);
  unsigned long long int *devDistCalcCountArr[numGPU];

  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    gpuErrchk(hipMalloc(&devDistCalcCountArr[i], sizeof(unsigned long long int)));
  }

  // variable initialization
  unsigned int hostConFlagArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hostConFlagArr[i] = 1;
  }

  unsigned int *hostConFlagPtrArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hostConFlagPtrArr[i] = &hostConFlagArr[i];
  }

  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo[numGPU];
  DTYPE *devPointData[numGPU];
  DTYPE *devPointLwrs[numGPU];

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hipSetDevice(i);

    // alloc dataset to GPU
    gpuErrchk(hipMalloc(&devPointInfo[i], sizeof(PointInfo)*(numPnts[i])));

    // copy input data to GPU
    gpuErrchk(hipMemcpy(devPointInfo[i],
                         pointInfo+(i*numPnt/numGPU),
                         (numPnts[i])*sizeof(PointInfo),
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&devPointData[i], sizeof(DTYPE) * numPnts[i] * numDim));

    gpuErrchk(hipMemcpy(devPointData[i],
                         pointData+((i*numPnt/numGPU) * numDim),
                         sizeof(DTYPE)*numPnts[i]*numDim,
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&devPointLwrs[i], sizeof(DTYPE) * numPnts[i] *
                         numGrp));

    gpuErrchk(hipMemcpy(devPointLwrs[i],
                         pointLwrs+((i*numPnt/numGPU) * numGrp),
                         sizeof(DTYPE)*numPnts[i]*numGrp,
                         hipMemcpyHostToDevice));
  }

  // store centroids on device
  CentInfo *devCentInfo[numGPU];
  DTYPE *devCentData[numGPU];
  DTYPE *devOldCentData[numGPU];

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));

    // alloc dataset and drift array to GPU
    gpuErrchk(hipMalloc(&devCentInfo[i], sizeof(CentInfo)*numCent));
    
    // alloc the old position data structure
    gpuErrchk(hipMalloc(&devOldCentData[i], sizeof(DTYPE) * numDim * numCent));

    // copy input data to GPU
    gpuErrchk(hipMemcpy(devCentInfo[i],
                         centInfo, sizeof(CentInfo)*numCent,
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&devCentData[i], sizeof(DTYPE)*numCent*numDim));
    gpuErrchk(hipMemcpy(devCentData[i],
                        centData, sizeof(DTYPE)*numCent*numDim,
                        hipMemcpyHostToDevice));
  }

  DTYPE *devMaxDriftArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devMaxDriftArr[i], sizeof(DTYPE) * numGrp);
  }

  // centroid calculation data
  DTYPE *devNewCentSum[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devNewCentSum[i], sizeof(DTYPE) * numCent * numDim);
  }

  DTYPE *devOldCentSum[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devOldCentSum[i], sizeof(DTYPE) * numCent * numDim);
  }

  unsigned int *devNewCentCount[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devNewCentCount[i], sizeof(unsigned int) * numCent);
  }

  unsigned int *devOldCentCount[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devOldCentCount[i], sizeof(unsigned int) * numCent);
  }

  unsigned int *devConFlagArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devConFlagArr[i], sizeof(unsigned int));
    gpuErrchk(hipMemcpy(devConFlagArr[i],
              hostConFlagPtrArr[i], sizeof(unsigned int),
              hipMemcpyHostToDevice));
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum[i],
                                              devOldCentSum[i],
                                              devNewCentCount[i],
                                              devOldCentCount[i],
                                              numCent,
                                              numDim);

  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr[i], numGrp);
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    // do single run of naive kmeans for initial centroid assignments
    initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo[i],
                                         devCentInfo[i],
                                         devPointData[i],
                                         devPointLwrs[i],
                                         devCentData[i],
                                         numPnts[i],
                                         numCent,
                                         numGrp,
                                         numDim,
                                         devDistCalcCountArr[i]);
  }

  CentInfo **allCentInfo = (CentInfo **)malloc(sizeof(CentInfo*)*numGPU);
  for (int i = 0; i < numGPU; i++)
  {
    allCentInfo[i] = (CentInfo *)malloc(sizeof(CentInfo)*numCent);
  }

  DTYPE **allCentData = (DTYPE **)malloc(sizeof(DTYPE*)*numGPU);
  for (int i = 0; i < numGPU; i++)
  {
    allCentData[i] = (DTYPE *)malloc(sizeof(DTYPE)*numCent*numDim);
  }

  CentInfo *newCentInfo = (CentInfo *)malloc(sizeof(CentInfo) * numCent);

  DTYPE *newCentData = (DTYPE *)malloc(sizeof(DTYPE) * numCent * numDim);
  for (int i = 0; i < numCent; i++)
  {
    for (int j = 0; j < numDim; j++)
    {
      newCentData[(i * numDim) + j] = 0;
    }
  }

  DTYPE *oldCentData = (DTYPE *)malloc(sizeof(DTYPE) * numCent * numDim);

  DTYPE *newMaxDriftArr;
  newMaxDriftArr=(DTYPE *)malloc(sizeof(DTYPE)*numGrp);
  for (int i = 0; i < numGrp; i++)
  {
    newMaxDriftArr[i] = 0.0;
  }

  unsigned int doesNotConverge = 1;

  // loop until convergence
  while(doesNotConverge && index < maxIter)
  {
    doesNotConverge = 0;

    for (int i = 0; i < numCent; i++)
    {
      newCentInfo[i].count = 0;
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      hostConFlagArr[i] = 0;
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(devConFlagArr[i],
                hostConFlagPtrArr[i], sizeof(unsigned int),
                hipMemcpyHostToDevice));
    }

    // clear maintained data on device
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr[i], numGrp);

    }


    // calculate data necessary to make new centroids
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                         devPointData[i],devOldCentSum[i],
                                         devNewCentSum[i],devOldCentCount[i],
                                         devNewCentCount[i],numPnts[i],numDim);

    }

    // make new centroids
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                             devCentData[i],devOldCentData[i],
                                             devOldCentSum[i],devNewCentSum[i],
                                             devMaxDriftArr[i],devOldCentCount[i],
                                             devNewCentCount[i],numCent,numDim);

    }

    if (numGPU > 1)
    {
      for (int i = 0; i < numGrp; i++)
      {
        newMaxDriftArr[i] = 0.0;
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
        gpuErrchk(hipSetDevice(i));
        gpuErrchk(hipMemcpy(allCentInfo[i],
                            devCentInfo[i], sizeof(CentInfo)*numCent,
                            hipMemcpyDeviceToHost));
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
        gpuErrchk(hipSetDevice(i));
        gpuErrchk(hipMemcpy(allCentData[i],
                            devCentData[i], sizeof(DTYPE)*numCent*numDim,
                            hipMemcpyDeviceToHost));
      }

      calcWeightedMeans(newCentInfo, allCentInfo, newCentData, oldCentData,
        allCentData, newMaxDriftArr, numCent, numGrp, numDim, numGPU);

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
          gpuErrchk(hipSetDevice(i));

          // copy input data to GPU
          gpuErrchk(hipMemcpy(devCentInfo[i],
                      newCentInfo, sizeof(cent)*numCent,
                                  hipMemcpyHostToDevice));
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
          gpuErrchk(hipSetDevice(i));

          // copy input data to GPU
          gpuErrchk(hipMemcpy(devCentData[i],
                      newCentData, sizeof(DTYPE)*numCent*numDim,
                                  hipMemcpyHostToDevice));
      }

      #pragma omp parallel for num_threads(numGPU)
      for (int i = 0; i < numGPU; i++)
      {
          gpuErrchk(hipSetDevice(i));
          gpuErrchk(hipMemcpy(devMaxDriftArr[i],
                       newMaxDriftArr, numGrp*sizeof(DTYPE),
                                  hipMemcpyHostToDevice));
      }
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      assignPointsSimple<<<NBLOCKS,BLOCKSIZE,grpLclSize>>>(devPointInfo[i],
                                                           devCentInfo[i],
                                                           devPointData[i],
                                                           devPointLwrs[i],
                                                           devCentData[i],
                                                           devMaxDriftArr[i],
                                                           numPnts[i],numCent,
                                                           numGrp,numDim,
                                                           devDistCalcCountArr[i]);

    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo[i],
                                           devConFlagArr[i],
                                           numPnts[i]);

    }

    index++;

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(hostConFlagPtrArr[i],
          devConFlagArr[i], sizeof(unsigned int),
                      hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < numGPU; i++)
    {
      if (hostConFlagArr[i])
      {
        doesNotConverge = 1;
      }
    }
  }
  
  hipDeviceSynchronize();
  
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    gpuErrchk(hipMemcpy(&hostDistCalcCountArr[i],
                devDistCalcCountArr[i], sizeof(unsigned long long int),
                            hipMemcpyDeviceToHost));
  }

  for (int i = 0; i < numGPU; i++)
  {
    //printf("hostDistCalcCountArr[%d]: %llu\n", i, hostDistCalcCountArr[i]);
    *hostDistCalcCount += hostDistCalcCountArr[i];
  }

  //printf("hostDistCalcCount: %llu\n", *hostDistCalcCount);
  
  *countPtr = *hostDistCalcCount;

  // calculate data necessary to make new centroids
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                        devPointData[i],devOldCentSum[i],
                                        devNewCentSum[i],devOldCentCount[i],
                                        devNewCentCount[i],numPnts[i],numDim);
  }

  // make new centroids
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                             devCentData[i],devOldCentData[i],
                                             devOldCentSum[i],devNewCentSum[i],
                                             devMaxDriftArr[i],devOldCentCount[i],
                                             devNewCentCount[i],numCent,numDim);
  }

  if (numGPU > 1)
  {
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(allCentInfo[i],
                          devCentInfo[i], sizeof(CentInfo)*numCent,
                          hipMemcpyDeviceToHost));
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(allCentData[i],
                          devCentData[i], sizeof(DTYPE)*numCent*numDim,
                          hipMemcpyDeviceToHost));
    }

    calcWeightedMeans(newCentInfo, allCentInfo, newCentData, oldCentData,
      allCentData, newMaxDriftArr, numCent, numGrp, numDim, numGPU);

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
        gpuErrchk(hipSetDevice(i));

        // copy input data to GPU
        gpuErrchk(hipMemcpy(devCentInfo[i],
                    newCentInfo, sizeof(cent)*numCent,
                                hipMemcpyHostToDevice));
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
        gpuErrchk(hipSetDevice(i));

        // copy input data to GPU
        gpuErrchk(hipMemcpy(devCentData[i],
                    newCentData, sizeof(DTYPE)*numCent*numDim,
                                hipMemcpyHostToDevice));
    }
  }

  hipDeviceSynchronize();

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));

    // copy finished clusters and points from device to host
    gpuErrchk(hipMemcpy(pointInfo+((i*numPnt/numGPU)),
                devPointInfo[i], sizeof(PointInfo)*numPnts[i], hipMemcpyDeviceToHost));
  }

  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData, devCentData[0],
                       sizeof(DTYPE)*numCent*numDim,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  for (int i = 0; i < numGPU; i++)
  {
    hipFree(devPointInfo[i]);
    hipFree(devPointData[i]);
    hipFree(devPointLwrs[i]);
    hipFree(devCentInfo[i]);
    hipFree(devCentData[i]);
    hipFree(devMaxDriftArr[i]);
    hipFree(devNewCentSum[i]);
    hipFree(devOldCentSum[i]);
    hipFree(devNewCentCount[i]);
    hipFree(devOldCentCount[i]);
    hipFree(devConFlagArr[i]);
  }

  free(allCentInfo);
  free(allCentData);
  free(newCentInfo);
  free(newCentData);
  free(oldCentData);
  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}

double startSuperOnGPU(PointInfo *pointInfo,
                     CentInfo *centInfo,
                     DTYPE *pointData,
                     DTYPE *centData,
                     const int numPnt,
                     const int numCent,
                     const int numDim,
                     const int maxIter,
                     unsigned int *ranIter,
                     unsigned long long int *countPtr)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);


  // group centroids
  for(int j = 0; j < numCent; j++)
  {
    centInfo[j].groupNum = 0;
  }

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt);
  for(int i = 0; i < numPnt; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, 1);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  DTYPE *devMaxDrift = NULL;
  hipMalloc(&devMaxDrift, sizeof(DTYPE));

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);
  
  DTYPE *devOldCentData = NULL;
  hipMalloc(&devOldCentData, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                         sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDrift, 1);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       1,
                                       numDim);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                           sizeof(unsigned int), hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDrift, 1);

    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDrift,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

    assignPointsSuper<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                              devPointData,devPointLwrs,
                                              devCentData,devMaxDrift,
                                              numPnt,numCent,1,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                         devConFlag,
                                         numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr, devConFlag,
                         sizeof(unsigned int), hipMemcpyDeviceToHost));
  }
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                             devCentData,devOldCentData,
                                             devOldCentSum,devNewCentSum,
                                             devMaxDrift,devOldCentCount,
                                             devNewCentCount,numCent,numDim);

  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                         sizeof(PointInfo)*numPnt, hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData, devCentData,
                         sizeof(DTYPE)*numDim*numCent, hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDrift);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);

  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}

double startLloydOnGPU(PointInfo *pointInfo,
                       CentInfo *centInfo,
                       DTYPE *pointData,
                       DTYPE *centData,
                       const int numPnt,
                       const int numCent,
                       const int numDim,
                       const int maxIter,
                       unsigned int *ranIter,
                       unsigned long long int *countPtr)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();
  
  

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;

  int index = 0;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;

  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                         sizeof(unsigned int), hipMemcpyHostToDevice));


  clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                                 devNewCentCount,
                                                 numCent,
                                                 numDim);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;

    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                         sizeof(unsigned int),hipMemcpyHostToDevice));

    assignPointsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                              devCentInfo,
                                              devPointData,
                                              devCentData,
                                              numPnt,
                                              numCent,
                                              numDim);

    clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                                   devNewCentCount,
                                                   numCent,
                                                   numDim);
    // calculate data necessary to make new centroids
    calcCentDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                              devPointData,
                                              devNewCentSum,
                                              devNewCentCount,
                                              numPnt,
                                              numDim);

    // make new centroids
    calcNewCentroidsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                                  devCentInfo,
                                                  devCentData,
                                                  devNewCentSum,
                                                  devNewCentCount,
                                                  numCent,
                                                  numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,devConFlag,numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr,devConFlag,
                           sizeof(unsigned int),hipMemcpyDeviceToHost));
  }
  hipDeviceSynchronize();
  *countPtr = (unsigned long long int)numPnt * 
    (unsigned long long int)numCent * (unsigned long long int)index;

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                         sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,
                         sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devNewCentSum);
  hipFree(devNewCentCount);
  hipFree(devConFlag);

  endTime = omp_get_wtime();
  return endTime - startTime;
}