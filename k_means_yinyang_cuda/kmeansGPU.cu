#include "hip/hip_runtime.h"
#include "kmeansGPU.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
 if (code != hipSuccess)
 {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
 }
}

/////////////////////////////////////////////
// Host functions for calling CUDA kernels //
/////////////////////////////////////////////

double startFullOnGPU(PointInfo *pointInfo, 
                    CentInfo *centInfo, 
                    DTYPE *pointData,
                    DTYPE *centData, 
                    const int numPnt, 
                    const int numCent,
                    const int numGrp, 
                    const int numDim, 
                    const int maxIter, 
                    unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;
  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;
  int oldPosSize = sizeof(DTYPE)*numDim*BLOCKSIZE;
  
  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);
  
  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, numGrp);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;
  
  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);


  DTYPE *devMaxDriftArr = NULL;
  hipMalloc(&devMaxDriftArr, sizeof(DTYPE) * numGrp);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr, 
                        sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo, 
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       numGrp,
                                       numDim);
  

  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;
    
    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr, 
                           sizeof(unsigned int),hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE, oldPosSize>>>(devPointInfo,devCentInfo,
                                                         devCentData,devOldCentSum,
                                                         devNewCentSum,devMaxDriftArr,
                                                         devOldCentCount,devNewCentCount,
                                                         numCent,numDim);

    assignPointsFull<<<NBLOCKS, BLOCKSIZE, grpLclSize>>>(devPointInfo,devCentInfo,
                                                         devPointData,devPointLwrs,
                                                         devCentData,devMaxDriftArr,
                                                         numPnt,numCent,numGrp,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,devConFlag,numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr,
        devConFlag, sizeof(unsigned int),
                    hipMemcpyDeviceToHost));
  }
  // calc final centroids (for matching results with lloyds)
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE, oldPosSize>>>(devPointInfo,devCentInfo,
                                                       devCentData,devOldCentSum,
                                                       devNewCentSum,devMaxDriftArr,
                                                       devOldCentCount,devNewCentCount,
                                                       numCent,numDim);
  
  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDriftArr);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);
  
  free(pointLwrs);
  
  endTime = omp_get_wtime();
  return endTime - startTime;
}


double startSimpleOnGPU(PointInfo *pointInfo,
                      CentInfo *centInfo,
                      DTYPE *pointData,
                      DTYPE *centData,
                      const int numPnt,
                      const int numCent,
                      const int numGrp,
                      const int numDim,
                      const int maxIter,
                      unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;
  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;
  int oldPosSize = sizeof(DTYPE)*numDim*BLOCKSIZE;
  
  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);


  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);
  
  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, numGrp);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;
  
  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  DTYPE *devMaxDriftArr = NULL;
  hipMalloc(&devMaxDriftArr, sizeof(DTYPE) * numGrp);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo, 
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       numGrp,
                                       numDim);

  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;
    
    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr, 
                           sizeof(unsigned int),hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr, numGrp);


    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE, oldPosSize>>>(devPointInfo,devCentInfo,
                                                         devCentData,devOldCentSum,
                                                         devNewCentSum,devMaxDriftArr,
                                                         devOldCentCount,devNewCentCount,
                                                         numCent,numDim);
    
    assignPointsSimple<<<NBLOCKS, BLOCKSIZE, grpLclSize>>>(devPointInfo,devCentInfo,
                                                           devPointData,devPointLwrs,
                                                           devCentData,devMaxDriftArr,
                                                           numPnt,numCent,numGrp,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                         devConFlag,
                                         numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr, devConFlag, 
                           sizeof(unsigned int), hipMemcpyDeviceToHost));
  }
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE, oldPosSize>>>(devPointInfo,devCentInfo,
                                                       devCentData,devOldCentSum,
                                                       devNewCentSum,devMaxDriftArr,
                                                       devOldCentCount,devNewCentCount,
                                                       numCent,numDim);
  
  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                       sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,
                       sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDriftArr);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);
  
  free(pointLwrs);
  
  endTime = omp_get_wtime();
  return endTime - startTime;
}


double startSimpleOnGPU(PointInfo *pointInfo,
                      CentInfo *centInfo,
                      DTYPE *pointData,
                      DTYPE *centData,
                      const int numPnt,
                      const int numCent,
                      const int numGrp,
                      const int numDim,
                      const int maxIter,
                      const int numGPU,
                      unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  int numPnts[numGPU];
  for (int i = 0; i < numGPU; i++)
  {
    if (numPnt % numGPU != 0 && i == numGPU-1)
    {
      numPnts[i] = (numPnt / numGPU) + (numPnt % numGPU);
    }

    else
    {
      numPnts[i] = numPnt / numGPU;
    }
    printf("    **Assigned %d points to GPU %d\n", numPnt, i);
    
  }

  // variable initialization
  unsigned int hostConFlagArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hostConFlagArr[i] = 1;
  }

  unsigned int *hostConFlagPtrArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hostConFlagPtrArr[i] = &hostConFlagArr[i];
  }

  int grpLclSize = sizeof(unsigned int)*numGrp*BLOCKSIZE;
  int oldPosSize = sizeof(DTYPE)*numDim*BLOCKSIZE;

  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // group centroids
  groupCent(centInfo, centData, numCent, numGrp, numDim);

  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt * numGrp);
  for(int i = 0; i < numPnt * numGrp; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo[numGPU];
  DTYPE *devPointData[numGPU];
  DTYPE *devPointLwrs[numGPU];

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    hipSetDevice(i);

    // alloc dataset to GPU
    gpuErrchk(hipMalloc(&devPointInfo[i], sizeof(PointInfo)*(numPnts[i])));
    printf("    **Allocated %d PointInfo Objects on GPU %d\n", numPnts[i], i);

    // copy input data to GPU
    gpuErrchk(hipMemcpy(devPointInfo[i],
                         pointInfo+(i*numPnt/numGPU),
                         (numPnts[i])*sizeof(PointInfo),
                         hipMemcpyHostToDevice));
    printf("    **Copied %d PointInfo Objects onto GPU %d\n", numPnts[i], i);

    gpuErrchk(hipMalloc(&devPointData[i], sizeof(DTYPE) * numPnts[i] * numDim));
    printf("    **Allocated %d DTYPE values (point data) on GPU %d\n", 
           numPnts[i] * numDim, i);

    // possible error here ???
    gpuErrchk(hipMemcpy(devPointData[i],
                         pointData+((i*numPnt/numGPU) * numDim),
                         sizeof(DTYPE)*numPnts[i]*numDim,
                         hipMemcpyHostToDevice));
    printf("    **Copied %d DTYPE values (point data) onto GPU %d\n", 
           numPnts[i] * numDim, i);

    gpuErrchk(hipMalloc(&devPointLwrs[i], sizeof(DTYPE) * numPnts[i] *
                         numGrp));
    printf("    **Allocated %d DTYPE values (lwr bounds) on GPU %d\n", 
           numPnts[i] * numGrp, i);

    gpuErrchk(hipMemcpy(devPointLwrs[i], 
                         pointLwrs+((i*numPnt/numGPU) * numGrp),
                         sizeof(DTYPE)*numPnts[i]*numGrp,
                         hipMemcpyHostToDevice));
    printf("    **Copied %d DTYPE values (lwr bounds) onto GPU %d\n", 
           numPnts[i] * numGrp, i);
  }

  // store centroids on device
  CentInfo *devCentInfo[numGPU];
  DTYPE *devCentData[numGPU];

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));

    // alloc dataset and drift array to GPU
    gpuErrchk(hipMalloc(&devCentInfo[i], sizeof(CentInfo)*numCent));
    printf("    **Allocated %d CentInfo Objects on GPU %d\n", numCent, i);

    // copy input data to GPU
    gpuErrchk(hipMemcpy(devCentInfo[i],
                         centInfo, sizeof(CentInfo)*numCent,
                         hipMemcpyHostToDevice));
    printf("    **Copied %d CentInfo Objects onto GPU %d\n", numCent, i);

    gpuErrchk(hipMalloc(&devCentData[i], sizeof(DTYPE)*numCent*numDim));
    printf("    **Allocated %d DTYPE values (centroid data) on GPU %d\n", 
           numCent * numDim, i);
    gpuErrchk(hipMemcpy(devCentData[i],
                        centInfo, sizeof(DTYPE)*numCent*numDim,
                        hipMemcpyHostToDevice));
    printf("    **Copied %d DTYPE values (centroid data) onto GPU %d\n", 
           numPnts[i] * numDim, i);
  }

  DTYPE *devMaxDriftArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devMaxDriftArr[i], sizeof(DTYPE) * numGrp);
    printf("    **Allocated %d DTYPE values (max drift array) on GPU %d\n", 
           numGrp, i);
  }

  // centroid calculation data
  DTYPE *devNewCentSum[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devNewCentSum[i], sizeof(DTYPE) * numCent * numDim);
    printf("    **Allocated %d DTYPE values (new centroid sums) on GPU %d\n", 
           numCent * numDim, i);
  }

  DTYPE *devOldCentSum[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devOldCentSum[i], sizeof(DTYPE) * numCent * numDim);
    printf("    **Allocated %d DTYPE values (old centroid sums) on GPU %d\n", 
           numCent * numDim, i);
  }

  unsigned int *devNewCentCount[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devNewCentCount[i], sizeof(unsigned int) * numCent);
    printf("    **Allocated %d unsigned int values (new centroid count) on GPU %d\n", numCent, i);
  }

  unsigned int *devOldCentCount[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devOldCentCount[i], sizeof(unsigned int) * numCent);
    printf("    **Allocated %d unsigned int values (old centroid count) on GPU %d\n", numCent, i);
  }

  unsigned int *devConFlagArr[numGPU];
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    hipMalloc(&devConFlagArr[i], sizeof(unsigned int));
    gpuErrchk(hipMemcpy(devConFlagArr[i],
              hostConFlagPtrArr[i], sizeof(unsigned int),
              hipMemcpyHostToDevice));
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    printf("    **Running clearCentCalcData on GPU %d\n", i);
    clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum[i],
                                              devOldCentSum[i],
                                              devNewCentCount[i],
                                              devOldCentCount[i],
                                              numCent,
                                              numDim);
    
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    printf("    **Running clearDriftArr on GPU %d\n", i);
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr[i], numGrp);
    
  }

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    printf("    **Running initRunKernel on GPU %d\n", i);
    // do single run of naive kmeans for initial centroid assignments
    initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo[i],
                                         devCentInfo[i],
                                         devPointData[i],
                                         devPointLwrs[i],
                                         devCentData[i],
                                         numPnts[i],
                                         numCent,
                                         numGrp,
                                         numDim);
  }



  unsigned int doesNotConverge = 1;

  // loop until convergence
  while(doesNotConverge && index < maxIter)
  {
    doesNotConverge = 0;

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      hostConFlagArr[i] = 0;
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(devConFlagArr[i],
                hostConFlagPtrArr[i], sizeof(unsigned int),
                hipMemcpyHostToDevice));
    }

    // clear maintained data on device
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      printf("    **Running clearDriftArr on GPU %d\n", i);
      clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr[i], numGrp);
      
    }


    // calculate data necessary to make new centroids
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      printf("    **Running calcCentData on GPU %d\n", i);
      calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                         devPointData[i],devOldCentSum[i],
                                         devNewCentSum[i],devOldCentCount[i],
                                         devNewCentCount[i],numPnts[i],numDim);
      
    }

    // make new centroids
    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      printf("    **Running calcNewCentroids on GPU %d\n", i);
      calcNewCentroids<<<NBLOCKS,BLOCKSIZE,oldPosSize>>>(devPointInfo[i],
                                                         devCentInfo[i],
                                                         devCentData[i],
                                                         devOldCentSum[i],
                                                         devNewCentSum[i],
                                                         devMaxDriftArr[i],
                                                         devOldCentCount[i],
                                                         devNewCentCount[i],
                                                         numCent,numDim);
      
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      hipSetDevice(i);
      printf("    **GPU %d is waiting at the barrier\n", i);
      hipDeviceSynchronize();
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      printf("    **Running assignPointsSimple on GPU %d\n", i);
      assignPointsSimple<<<NBLOCKS,BLOCKSIZE,grpLclSize>>>(devPointInfo[i],
                                                           devCentInfo[i],
                                                           devPointData[i],
                                                           devPointLwrs[i],
                                                           devCentData[i],
                                                           devMaxDriftArr[i],
                                                           numPnts[i],numCent,
                                                           numGrp,numDim);
      
    }

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      printf("  **Running checkConverge on GPU %d\n", i);
      checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo[i],
                                           devConFlagArr[i],
                                           numPnts[i]);
      
    }

    index++;

    #pragma omp parallel for num_threads(numGPU)
    for (int i = 0; i < numGPU; i++)
    {
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(hostConFlagPtrArr[i],
          devConFlagArr[i], sizeof(unsigned int),
                      hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < numGPU; i++)
    {
      if (hostConFlagArr[i])
      {
        doesNotConverge = 1;
      }
    }
  }

  // calculate data necessary to make new centroids
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    printf("    **Running final calcCentData on GPU %d\n", i);
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo[i],devCentInfo[i],
                                        devPointData[i],devOldCentSum[i],
                                        devNewCentSum[i],devOldCentCount[i],
                                        devNewCentCount[i],numPnts[i],numDim);
  }

  // make new centroids
  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    printf("    **Running final calcNewCent on GPU %d\n", i);
    calcNewCentroids<<<NBLOCKS,BLOCKSIZE,oldPosSize>>>(devPointInfo[i],
                                                       devCentInfo[i],
                                                       devCentData[i],
                                                       devOldCentSum[i],
                                                       devNewCentSum[i],
                                                       devMaxDriftArr[i],
                                                       devOldCentCount[i],
                                                       devNewCentCount[i],
                                                       numCent,numDim);
  }

  hipDeviceSynchronize();

  #pragma omp parallel for num_threads(numGPU)
  for (int i = 0; i < numGPU; i++)
  {
    gpuErrchk(hipSetDevice(i));
    
    // copy finished clusters and points from device to host
    gpuErrchk(hipMemcpy(pointInfo+((i*numPnt/numGPU)),
                devPointInfo[i], sizeof(PointInfo)*numPnts[i], hipMemcpyDeviceToHost));
  }

  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData[0],
                       sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  for (int i = 0; i < numGPU; i++)
  {
    printf("    **Freeing data on GPU %d\n", i);
    hipFree(devPointInfo[i]);
    hipFree(devPointData[i]);
    hipFree(devPointLwrs[i]);
    hipFree(devCentInfo[i]);
    hipFree(devCentData[i]);
    hipFree(devMaxDriftArr[i]);
    hipFree(devNewCentSum[i]);
    hipFree(devOldCentSum[i]);
    hipFree(devNewCentCount[i]);
    hipFree(devOldCentCount[i]);
    hipFree(devConFlagArr[i]);
  }


  free(pointLwrs);

  endTime = omp_get_wtime();
  return endTime - startTime;
}

double startSuperOnGPU(PointInfo *pointInfo,
                     CentInfo *centInfo,
                     DTYPE *pointData, 
                     DTYPE *centData,
                     const int numPnt,
                     const int numCent,
                     const int numDim,
                     const int maxIter,
                     unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;
  int oldPosSize = sizeof(DTYPE)*numDim*BLOCKSIZE;
  
  int index = 1;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);


  // group centroids
  for(int j = 0; j < numCent; j++)
  {
    centInfo[j].groupNum = 0;
  }
  
  // create lower bound data on host
  DTYPE *pointLwrs = (DTYPE *)malloc(sizeof(DTYPE) * numPnt);
  for(int i = 0; i < numPnt; i++)
  {
    pointLwrs[i] = INFINITY;
  }

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;
  DTYPE *devPointLwrs;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);
  devPointLwrs = storeDataOnGPU(pointLwrs, numPnt, 1);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;
  
  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  DTYPE *devMaxDrift = NULL;
  hipMalloc(&devMaxDrift, sizeof(DTYPE));

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  DTYPE *devOldCentSum = NULL;
  hipMalloc(&devOldCentSum, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devOldCentCount = NULL;
  hipMalloc(&devOldCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                         sizeof(unsigned int),hipMemcpyHostToDevice));


  clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                            devOldCentSum,
                                            devNewCentCount,
                                            devOldCentCount,
                                            numCent,
                                            numDim);

  clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDrift, 1);

  // do single run of naive kmeans for initial centroid assignments
  initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo, 
                                       devCentInfo,
                                       devPointData,
                                       devPointLwrs,
                                       devCentData,
                                       numPnt,
                                       numCent,
                                       1,
                                       numDim);
  

  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;
    
    gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                           sizeof(unsigned int), hipMemcpyHostToDevice));

    // clear maintained data on device
    clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDrift, 1);

    // calculate data necessary to make new centroids
    calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                         devPointData,devOldCentSum,
                                         devNewCentSum,devOldCentCount,
                                         devNewCentCount,numPnt,numDim);

    // make new centroids
    calcNewCentroids<<<NBLOCKS, BLOCKSIZE, oldPosSize>>>(devPointInfo,devCentInfo,
                                                         devCentData,devOldCentSum,
                                                         devNewCentSum,devMaxDrift,
                                                         devOldCentCount,devNewCentCount,
                                                         numCent,numDim);
    
    assignPointsSuper<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                              devPointData,devPointLwrs,
                                              devCentData,devMaxDrift,
                                              numPnt,numCent,1,numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,
                                         devConFlag,
                                         numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr, devConFlag,
                         sizeof(unsigned int), hipMemcpyDeviceToHost));
  }
  calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,devCentInfo,
                                       devPointData,devOldCentSum,
                                       devNewCentSum,devOldCentCount,
                                       devNewCentCount,numPnt,numDim);

  // make new centroids
  calcNewCentroids<<<NBLOCKS, BLOCKSIZE, oldPosSize>>>(devPointInfo,devCentInfo,
                                                       devCentData,devOldCentSum,
                                                       devNewCentSum,devMaxDrift,
                                                       devOldCentCount,devNewCentCount,
                                                       numCent,numDim);
  
  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                         sizeof(PointInfo)*numPnt, hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData, devCentData,
                         sizeof(DTYPE)*numDim*numCent, hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devPointLwrs);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devMaxDrift);
  hipFree(devNewCentSum);
  hipFree(devOldCentSum);
  hipFree(devNewCentCount);
  hipFree(devOldCentCount);
  hipFree(devConFlag);
  
  free(pointLwrs);
  
  endTime = omp_get_wtime();
  return endTime - startTime;
}

double startLloydOnGPU(PointInfo *pointInfo,
                     CentInfo *centInfo,
                     DTYPE *pointData, 
                     DTYPE *centData,
                     const int numPnt,
                     const int numCent,
                     const int numDim,
                     const int maxIter,
                     unsigned int *ranIter)
{

  // start timer
  double startTime, endTime;
  startTime = omp_get_wtime();

  // variable initialization

  unsigned int hostConFlag = 1;

  unsigned int *hostConFlagPtr = &hostConFlag;

  int index = 0;

  unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);

  // store dataset on device
  PointInfo *devPointInfo;
  DTYPE *devPointData;

  devPointInfo = storePointInfoOnGPU(pointInfo, numPnt);
  devPointData = storeDataOnGPU(pointData, numPnt, numDim);

  // store centroids on device
  CentInfo *devCentInfo;
  DTYPE *devCentData;
  
  devCentInfo = storeCentInfoOnGPU(centInfo, numCent);
  devCentData = storeDataOnGPU(centData, numCent, numDim);

  // centroid calculation data
  DTYPE *devNewCentSum = NULL;
  hipMalloc(&devNewCentSum, sizeof(DTYPE) * numCent * numDim);

  unsigned int *devNewCentCount = NULL;
  hipMalloc(&devNewCentCount, sizeof(unsigned int) * numCent);

  unsigned int *devConFlag = NULL;
  hipMalloc(&devConFlag, sizeof(unsigned int));

  gpuErrchk(hipMemcpy(devConFlag, hostConFlagPtr,
                         sizeof(unsigned int), hipMemcpyHostToDevice));


  clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                                 devNewCentCount,
                                                 numCent,
                                                 numDim);


  // loop until convergence
  while(hostConFlag && index < maxIter)
  {
    hostConFlag = 0;
    
    gpuErrchk(hipMemcpy(devConFlag,hostConFlagPtr,
                         sizeof(unsigned int),hipMemcpyHostToDevice));
    
    assignPointsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                              devCentInfo,
                                              devPointData,
                                              devCentData,
                                              numPnt,
                                              numCent,
                                              numDim);

    clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
                                                   devNewCentCount,
                                                   numCent,
                                                   numDim);
    // calculate data necessary to make new centroids
    calcCentDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                              devPointData,
                                              devNewCentSum,
                                              devNewCentCount,
                                              numPnt,
                                              numDim);

    // make new centroids
    calcNewCentroidsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devPointInfo,
                                                  devCentInfo,
                                                  devCentData,
                                                  devNewCentSum,
                                                  devNewCentCount,
                                                  numCent,
                                                  numDim);

    checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devPointInfo,devConFlag,numPnt);
    index++;
    gpuErrchk(hipMemcpy(hostConFlagPtr,devConFlag,
                           sizeof(unsigned int),hipMemcpyDeviceToHost));
  }
  hipDeviceSynchronize();

  // only need the point info for assignments
  gpuErrchk(hipMemcpy(pointInfo, devPointInfo,
                         sizeof(PointInfo)*numPnt,hipMemcpyDeviceToHost));
  // and the final centroid positions
  gpuErrchk(hipMemcpy(centData,devCentData,
                         sizeof(DTYPE)*numDim*numCent,hipMemcpyDeviceToHost));

  *ranIter = index;

  // clean up, return
  hipFree(devPointInfo);
  hipFree(devPointData);
  hipFree(devCentInfo);
  hipFree(devCentData);
  hipFree(devNewCentSum);
  hipFree(devNewCentCount);
  hipFree(devConFlag);
  
  endTime = omp_get_wtime();
  return endTime - startTime;
}

PointInfo *storePointInfoOnGPU(PointInfo *pointInfo, 
                               const int numPnt)
{
  PointInfo *devPointInfo = NULL;
  gpuErrchk(hipMalloc(&devPointInfo, sizeof(PointInfo)*numPnt));
  gpuErrchk(hipMemcpy(devPointInfo, pointInfo, 
                         sizeof(PointInfo)*numPnt, hipMemcpyHostToDevice));
  return devPointInfo;
}

CentInfo *storeCentInfoOnGPU(CentInfo *centInfo, 
                             const int numCent)
{
  CentInfo *devCentInfo = NULL;
  gpuErrchk(hipMalloc(&devCentInfo, sizeof(CentInfo) * numCent));
  gpuErrchk(hipMemcpy(devCentInfo, centInfo, 
                         sizeof(CentInfo)*numCent, hipMemcpyHostToDevice));
  return devCentInfo;
}

DTYPE *storeDataOnGPU(DTYPE *data,
                      const int numVec,
                      const int numFeat)
{
  DTYPE *devData = NULL;
  gpuErrchk(hipMalloc(&devData, sizeof(DTYPE) * numVec * numFeat));
  gpuErrchk(hipMemcpy(devData, data, 
                         sizeof(DTYPE)*numVec*numFeat, hipMemcpyHostToDevice));
  return devData;
}

void warmupGPU()
{
  unsigned int *dev_tmp;
  unsigned int *tmp;
  tmp = (unsigned int*)malloc(sizeof(unsigned int));
  *tmp = 0;
  hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));

  warmup<<<1,256>>>(dev_tmp);

  hipMemcpy(tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(dev_tmp);
}
