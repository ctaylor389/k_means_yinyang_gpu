#include "hip/hip_runtime.h"
#include "GPU.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void warmupGPU()
{
	unsigned int *dev_tmp;
	unsigned int *tmp;
	tmp = (unsigned int*)malloc(sizeof(unsigned int));
	*tmp = 0;
	hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));

	warmup<<<1,256>>>(dev_tmp);

	hipMemcpy(tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	
	hipFree(dev_tmp);
}



point *storeDatasetOnGPU(point *dataset,
						 unsigned long int numPnt)
{
	point * dev_inputData = NULL;

	// alloc dataset to GPU
	gpuErrchk(hipMalloc(&dev_inputData, numPnt*sizeof(point)));

	// copy input data to GPU
	gpuErrchk(hipMemcpy(dev_inputData, 
				dataset, numPnt*sizeof(point), 
							hipMemcpyHostToDevice));

	unsigned int NBLOCKS = ceil(numPnt*1.0/BLOCKSIZE*1.0);
	return dev_inputData;

}

cent *storeCentDataOnGPU(cent *centDataset,
						 const unsigned int numCent)
{
	cent * dev_centInputData = NULL;


	// alloc dataset and drift array to GPU
	gpuErrchk(hipMalloc(&dev_centInputData, numCent*sizeof(cent)));
	

	// copy input data to GPU
	gpuErrchk(hipMemcpy(dev_centInputData, 
				centDataset, numCent*sizeof(cent), 
							hipMemcpyHostToDevice));


	unsigned int NBLOCKS = ceil(numCent*1.0/BLOCKSIZE*1.0);
	
	return dev_centInputData;
}


/*
function containing master loop that calls yinyang kernels
*/
int startYinYangOnGPU(point *hostDataset,
					  cent *hostCentDataset,
					  unsigned long long int *hostDistCalcCount,
					  double *yinStartTime,
					  double *yinEndTime,
					  unsigned int *ranIter)
{

	// start timer
	*yinStartTime = omp_get_wtime();




	// variable initialization

	unsigned int hostConFlag = 1;

	unsigned int *hostConFlagPtr = &hostConFlag;
	
	unsigned int index = 0;

	unsigned int NBLOCKS = ceil(NPOINT*1.0/BLOCKSIZE*1.0);


	// group centroids
	groupCent(hostCentDataset, NCLUST, NGROUP, NDIM);

	// store dataset on device
	point *devDataset;

	devDataset = storeDatasetOnGPU(hostDataset, NPOINT);

	// store centroids on device
	cent *devCentDataset;

	devCentDataset = storeCentDataOnGPU(hostCentDataset,
										NCLUST);

	// allocate device-only data
	unsigned long long int *devDistCalcCount = NULL;

	gpuErrchk(hipMalloc(&devDistCalcCount, sizeof(unsigned long long int)));
	
	gpuErrchk(hipMemcpy(devDistCalcCount, 
			hostDistCalcCount, sizeof(unsigned long long int), 
						hipMemcpyHostToDevice));


	double *devMaxDriftArr = NULL;
	hipMalloc(&devMaxDriftArr, NGROUP*sizeof(double));

	// centroid calculation data
	struct vector *devNewCentSum = NULL;
	hipMalloc(&devNewCentSum, NCLUST*sizeof(vector));

	struct vector *devOldCentSum = NULL;
	hipMalloc(&devOldCentSum, NCLUST*sizeof(vector));

	unsigned int *devNewCentCount = NULL;
	hipMalloc(&devNewCentCount, NCLUST*sizeof(unsigned int));

	unsigned int *devOldCentCount = NULL;
	hipMalloc(&devOldCentCount, NCLUST*sizeof(unsigned int));

	unsigned int *devConFlag = NULL;
	hipMalloc(&devConFlag, sizeof(unsigned int));

	gpuErrchk(hipMemcpy(devConFlag, 
			hostConFlagPtr, sizeof(unsigned int), 
						hipMemcpyHostToDevice));

	

	clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
											  devOldCentSum,
											  devNewCentCount,
											  devOldCentCount);
											  
	clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr);



	// do single run of naive kmeans for initial centroid assignments	
	initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devDataset, 
										 devCentDataset,
										 devDistCalcCount);


	// loop until convergence
	while(hostConFlag && index < MAXITER)
	{
		hostConFlag = 0;
		
		gpuErrchk(hipMemcpy(devConFlag, 
			hostConFlagPtr, sizeof(unsigned int), 
						hipMemcpyHostToDevice));	

						
		// clear maintained data on device
		clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr);
		
		clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
											  	devOldCentSum,
											  	devNewCentCount,
											  	devOldCentCount);


		// calculate data necessary to make new centroids
		calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
					 						 devCentDataset,
					 						 devOldCentSum,
					 						 devNewCentSum,
					 						 devOldCentCount,
					 						 devNewCentCount);

		// make new centroids
		calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
						 						 devCentDataset,
						 						 devMaxDriftArr,
						 						 devOldCentSum,
						 						 devNewCentSum,
						 						 devOldCentCount,
						 						 devNewCentCount);

						 						 
		// update point assignments via assignPointsernel
		assignPointsYinyang<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
												    devCentDataset,
												    devMaxDriftArr,
												    devDistCalcCount);

		checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devDataset, devConFlag);					   
		index++;
		gpuErrchk(hipMemcpy(hostConFlagPtr, 
			devConFlag, sizeof(unsigned int), 
						hipMemcpyDeviceToHost));
	}

	hipDeviceSynchronize();


	//printf("\ntotal iter: %d\n", index);
	
	// copy finished clusters and points from device to host
	gpuErrchk(hipMemcpy(hostDataset,
				devDataset, NPOINT*sizeof(struct point),
							hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hostCentDataset,
				devCentDataset, NCLUST*sizeof(struct cent),
							hipMemcpyDeviceToHost));

	*yinEndTime = omp_get_wtime();

	*ranIter = index + 1;

	// clean up, return
	hipFree(devDistCalcCount);
	hipFree(devMaxDriftArr);
	hipFree(devNewCentSum);
	hipFree(devOldCentSum);
	hipFree(devNewCentCount);
	hipFree(devOldCentCount);
	hipFree(devDataset);
	hipFree(devCentDataset);
	hipFree(devConFlag);

	
	return 0;
}




int startLloydOnGPU(point *hostDataset,
					cent *hostCentDataset,
					double *lloydStartTime,
					double *lloydEndTime,
					unsigned int *ranIter)
{


	// start timer
	*lloydStartTime = omp_get_wtime();


	unsigned int hostConFlag = 1;

	unsigned int *hostConFlagPtr = &hostConFlag;
	
	unsigned int index = 0;


	// store dataset on device
	point *devDataset;

	devDataset = storeDatasetOnGPU(hostDataset, NPOINT);

	// store centroids on device
	cent *devCentDataset;

	devCentDataset = storeCentDataOnGPU(hostCentDataset,
										NCLUST);


	unsigned int NBLOCKS = ceil(NPOINT*1.0/BLOCKSIZE*1.0);

	unsigned int *devNewCentCount = NULL;
	hipMalloc(&devNewCentCount, NCLUST * sizeof(unsigned int));

	struct vector *devNewCentSum = NULL;
	hipMalloc(&devNewCentSum, NCLUST * sizeof(vector));



	unsigned int *devConFlag = NULL;
	hipMalloc(&devConFlag, sizeof(unsigned int));

	gpuErrchk(hipMemcpy(devConFlag, 
			hostConFlagPtr, sizeof(unsigned int), 
						hipMemcpyHostToDevice));
	

	clearCentCalcDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
											 	   devNewCentCount);

	// master loop for maxIter runs
	while(hostConFlag && index < MAXITER)
	{
		hostConFlag = 0;
		
		gpuErrchk(hipMemcpy(devConFlag, 
			hostConFlagPtr, sizeof(unsigned int), 
						hipMemcpyHostToDevice));
		
		// update point assignments via assignPointsernel
		assignPointsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
												   devCentDataset);


		calcCentDataLloyd<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
					 						 devCentDataset,
					 						 devNewCentSum,
					 						 devNewCentCount);
	
		calcNewCentroidsLloyd<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
						 						 devCentDataset,
						 						 devNewCentSum,
						 						 devNewCentCount);


		checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devDataset, devConFlag);					   
		
		index++;
		gpuErrchk(hipMemcpy(hostConFlagPtr, 
			devConFlag, sizeof(unsigned int), 
						hipMemcpyDeviceToHost));
	}


	hipDeviceSynchronize();


	//printf("\ntotal iter: %d\n", index);
	
	// copy assigned data from device to host
	gpuErrchk(hipMemcpy(hostDataset,
				devDataset, NPOINT * sizeof(point),
							hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hostCentDataset,
				devCentDataset, NCLUST * sizeof(cent),
							hipMemcpyDeviceToHost));
	*lloydEndTime = omp_get_wtime();

	*ranIter = index;

	hipFree(devDataset);
	hipFree(devCentDataset);
	hipFree(devNewCentCount);
	hipFree(devNewCentSum);
	hipFree(devConFlag);
	return 0;
}



/*
function containing master loop that calls yinyang kernels
*/
int startHamerlyOnGPU(point *hostDataset,
					  cent *hostCentDataset,
					  unsigned long long int *hostDistCalcCount,
					  double *hamStartTime,
					  double *hamEndTime,
					  unsigned int *ranIter)
{

	// start timer
	*hamStartTime = omp_get_wtime();

	unsigned int hostConFlag = 1;

	unsigned int *hostConFlagPtr = &hostConFlag;
	
	unsigned int index = 0;
	

	unsigned int clustIndex;

	unsigned int NBLOCKS = ceil(NPOINT*1.0/BLOCKSIZE*1.0);

	// assign all centroids to 0 group
	for(clustIndex = 0; clustIndex < NCLUST; clustIndex++)
	{
		hostCentDataset[clustIndex].groupNum = 0;
	}

	// store dataset on device
	point *devDataset;

	devDataset = storeDatasetOnGPU(hostDataset, NPOINT);

	// store centroids on device
	cent *devCentDataset;

	devCentDataset = storeCentDataOnGPU(hostCentDataset,
										NCLUST);

	// allocate a count on the GPU
	unsigned long long int *devDistCalcCount = NULL;

	gpuErrchk(hipMalloc(&devDistCalcCount, sizeof(unsigned long long int)));

	gpuErrchk(hipMemcpy(devDistCalcCount, 
			hostDistCalcCount, sizeof(unsigned long long int), 
						hipMemcpyHostToDevice));

	double *devMaxDriftArr = NULL;
	hipMalloc(&devMaxDriftArr, NGROUP*sizeof(double));

	// centroid calculation data
	struct vector *devNewCentSum = NULL;
	hipMalloc(&devNewCentSum, NCLUST*sizeof(vector));

	struct vector *devOldCentSum = NULL;
	hipMalloc(&devOldCentSum, NCLUST*sizeof(vector));

	unsigned int *devNewCentCount = NULL;
	hipMalloc(&devNewCentCount, NCLUST*sizeof(unsigned int));

	unsigned int *devOldCentCount = NULL;
	hipMalloc(&devOldCentCount, NCLUST*sizeof(unsigned int));


	unsigned int *devConFlag = NULL;
	hipMalloc(&devConFlag, sizeof(unsigned int));

	gpuErrchk(hipMemcpy(devConFlag, 
			hostConFlagPtr, sizeof(unsigned int), 
						hipMemcpyHostToDevice));
	


	clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
											  	devOldCentSum,
											  	devNewCentCount,
											  	devOldCentCount);



											  
	clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr);

	// do single run of naive kmeans for initial centroid assignments	
	initRunKernel<<<NBLOCKS,BLOCKSIZE>>>(devDataset, 
										 devCentDataset,
										 devDistCalcCount);



	// master loop for maxIter runs
	while(hostConFlag && index < MAXITER)
	{

		hostConFlag = 0;

		gpuErrchk(hipMemcpy(devConFlag, 
			hostConFlagPtr, sizeof(unsigned int), 
						hipMemcpyHostToDevice));
	
		clearDriftArr<<<NBLOCKS, BLOCKSIZE>>>(devMaxDriftArr);

		clearCentCalcData<<<NBLOCKS, BLOCKSIZE>>>(devNewCentSum,
												  devOldCentSum,
												  devNewCentCount,
												  devOldCentCount);


		calcCentData<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
					 						 devCentDataset,
					 						 devOldCentSum,
					 						 devNewCentSum,
					 						 devOldCentCount,
					 						 devNewCentCount);
	
		calcNewCentroids<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
						 						 devCentDataset,
						 						 devMaxDriftArr,
						 						 devOldCentSum,
						 						 devNewCentSum,
						 						 devOldCentCount,
						 						 devNewCentCount);

		
		// update point assignments via assignPointsernel
		assignPointsHamerly<<<NBLOCKS, BLOCKSIZE>>>(devDataset,
												    devCentDataset,
												    devMaxDriftArr,
												   	devDistCalcCount);

		checkConverge<<<NBLOCKS,BLOCKSIZE>>>(devDataset, devConFlag);

		index++;
		gpuErrchk(hipMemcpy(hostConFlagPtr, 
			devConFlag, sizeof(unsigned int), 
						hipMemcpyDeviceToHost));


	}

	hipDeviceSynchronize();
	
	

	// copy assigned data from device to host
	gpuErrchk(hipMemcpy(hostDataset,
				devDataset, NPOINT*sizeof(point),
							hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(hostCentDataset,
				devCentDataset, NCLUST*sizeof(cent),
							hipMemcpyDeviceToHost));


	*hamEndTime = omp_get_wtime();


	*ranIter = index + 1;


	hipFree(devDataset);
	hipFree(devCentDataset);
	hipFree(devMaxDriftArr);
	hipFree(devNewCentSum);
	hipFree(devOldCentSum);
	hipFree(devNewCentCount);
	hipFree(devOldCentCount);
	hipFree(devConFlag);
	
	return 0;
}



